#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void cudaUpsweep(long long N, long long two_d, long long two_dplus_1, int* data, long long roundedN) {
    long long idx = blockIdx.x * blockDim.x + threadIdx.x; // Thread index
    long long i = idx * two_dplus_1; // Actual element index

    if(i + two_dplus_1 - 1 >= roundedN) return;

    data[i + two_dplus_1 - 1] += data[i + two_d - 1];
}

__global__ void cudaDownsweep(long long N, long long two_d, long long two_dplus_1, int* data, long long pow2N) {
    long long idx = blockIdx.x * blockDim.x + threadIdx.x; // Thread index
    long long i = idx * two_dplus_1; // Actual element index

    if(i + two_dplus_1 - 1 >= pow2N) return;

    int t = data[i + two_d - 1];
    data[i + two_d - 1] = data[i + two_dplus_1 - 1];
    data[i + two_dplus_1 - 1] += t;
}

__global__ void cudaMiddleStep(int* data, long long pow2N) {
    if(blockIdx.x * blockDim.x + threadIdx.x == 0) {
        data[pow2N - 1] = 0;
    }
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    // Upsweep
    long long roundedN = nextPow2(N);

    for(int two_d = 1; two_d <= roundedN / 2; two_d *= 2) {
	    int two_dplus1 = 2 * two_d;

        int ops = roundedN / two_dplus1;
        int numBlocks = (ops + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	    cudaUpsweep<<<numBlocks, THREADS_PER_BLOCK>>>(N, two_d, two_dplus1, result, roundedN);
        hipDeviceSynchronize();
    }

    cudaMiddleStep<<<1, THREADS_PER_BLOCK>>>(result, roundedN);
    hipDeviceSynchronize();

    // Downsweep
    for(int two_d = roundedN / 2; two_d >= 1; two_d /= 2) {
	    int two_dplus1 = 2 * two_d;
        
        int ops = roundedN / two_dplus1;
        int numBlocks = (ops + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        cudaDownsweep<<<numBlocks, THREADS_PER_BLOCK>>>(N, two_d, two_dplus1, result, roundedN);
        hipDeviceSynchronize();
    }
}

//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

__global__ void pair_and_compare(int* input_data, int* positions_mask, int length){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; //establish thread identity 
    if (idx >= length - 1) return; //is it possible that the input_data size is smalled than the length? 
    if (input_data[idx] == input_data[idx + 1]) {
        positions_mask[idx] = 1;
    } 
    else {
        positions_mask[idx] = 0;
    }
}

__global__ void cuda_identify_transition_points(int* prefix_sum_array, int* result, int length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Unique thread index

    if (idx >= length - 1) return; // Ensure we stay within bounds for idx + 1

    // Check if we have a transition point
    if (prefix_sum_array[idx + 1] == prefix_sum_array[idx] + 1) {
        result[prefix_sum_array[idx]] = idx;
    }
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output, int *positions_mask) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    // Step 1: Call pair_and_compare to populate positions_mask
    int numBlocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    pair_and_compare<<<numBlocks, THREADS_PER_BLOCK>>>(device_input, positions_mask, length);
    hipDeviceSynchronize();

    // Step 2: Perform an exclusive scan on positions_mask (keeping it on the GPU)
    exclusive_scan(positions_mask, length, positions_mask); // Result remains in positions_mask
    // hipDeviceSynchronize(); //do we need this? 

    // Step 3: Identify transition points using cuda_identify_transition_points
    cuda_identify_transition_points<<<numBlocks, THREADS_PER_BLOCK>>>(positions_mask, device_output, length);
    hipDeviceSynchronize();

    // Step 4: Get the last element of positions_mask as output length using Thrust
    int output_length = dev_mask_ptr[length - 1];

    // Return the count of repeated elements
    //return output_length;

    return 0; 
}




//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int *positions_mask; //positions[i] = 1 if the device_input[i] = device input[i+1] 
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&positions_mask, rounded_length * sizeof(int)); //allocate memory for positions_mask
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
